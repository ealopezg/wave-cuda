#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <getopt.h>

#include "wave_cuda.cuh"

/**
 * @brief Estructura donde se guardan los parámetros de entrada
 *
 */
typedef struct
{
    int grid_size;  // Tamaño de la matriz
    int steps;      // Cantidad de iteraciones
    int x;
    int y;
    char *filename; // Nombre archivo de salida
} options_t;

/**
 * @brief Crea una matriz de 3 dimensiones , N*N*3, donde la tercera dimensión corresponde a los pasos anteriores (0: t,1: t-1,2: t-2)
 *
 * @param N Tamaño de la matriz
 * @return float*** Matriz
 */
float* initializeGrid(int N){
    float* grid = (float*)malloc(N*N*3*sizeof(float));
    for(int i = 0;i <N*N*3;i++){
        grid[i] = 0;
    }
    return grid;
}


// Calcula la iteración t en cada caso
/**
 * @brief Calcula la iteración en cada caso
 *
 * @param grid Matriz de flotantes de NxN
 * @param N Tamaño de la matriz
 * @param i Posición fila
 * @param j Posición Columna
 * @param t Número Iteración
 */
__device__ void schroedinger_(float *grid, int N, int i, int j, int t)
{

    switch (t)
    {
    case 0:
        // Tiene que ser N-1 porque o sino el impulso inicial no estaría centrado
        if ((0.4 * (N - 1) < i) && (i < 0.6 * (N - 1)) && (0.4 * (N - 1) < j) && (j < 0.6 * (N - 1)))
        {

            grid[3 * (N * i + j) + 0] = 20;
        }
        else
        {
            grid[3 * (N * i + j) + 0] = 0;
        }
        break;
    case 1:
        grid[3*(N*i +j)+ 0] = grid[3*(N*i +j)+ 1] + (0.00125 * (grid[3*(N*(i+1) +j)+ 1] + grid[3*(N*(i-1) +j)+ 1] + grid[3*(N*i +(j-1))+ 1] + grid[3*(N*i +(j+1))+ 1] - (4 * grid[3*(N*i +j)+ 1])));
        break;
    default:
        grid[3*(N*i +j)+ 0] = (2 * grid[3*(N*i +j)+ 1]) - grid[3*(N*i +j)+ 2] + (0.0025 * (grid[3*(N*(i+1) +j)+ 1] + grid[3*(N*(i-1) +j)+ 1] + grid[3*(N*i +(j-1))+ 1] + grid[3*(N*i +(j+1))+ 1] - (4 * grid[3*(N*i +j)+ 1])));
        break;
    }
}

/**
 * @brief Calcula la difusión de una onda según la equación de Schroendinger
 *
 * @param grid Matriz de flotantes de NxN
 * @param N Tamaño de la matriz
 * @param T Número de iteración de salida
 */
__global__ void schroedinger(float* grid, int N, int T, int t){
    /*for(int t = 0; t <= T; t++){
        for(int i = 1;i < N-1;i++){
            for(int j = 1;j <N-1;j++){
                schroedinger_(grid,N,i,j,t);
            }
        }
        if(t != T){
            for (int i = 1; i < N - 1; i++)
            {
                for (int j = 1; j < N - 1; j++)
                {
                    grid[3*(N*i +j)+ 2] = grid[3*(N*i +j)+ 1];
                    grid[3*(N*i +j)+ 1] = grid[3*(N*i +j)+ 0];
                }
            }
        }
        
    }*/
    int blocksize = blockDim.y * blockDim.x; // number of threads in a TB
    //printf("tamano bloque: %d\n", blocksize);
    int blockId = gridDim.x * blockIdx.y + blockIdx.x; // unique block Id
    //printf("blockId: %d\n", blockId);
    //int tid = blockId * blocksize + blockDim.x*threadIdx.y + threadIdx.x; // global tid
    int tIdX = blockDim.x*blockIdx.x + threadIdx.x;
    int tIdY = blockDim.y*blockIdx.y + threadIdx.y;
    
    schroedinger_(grid,N,tIdX,tIdY,t);
    //printf("t: %d tIdX: %d tIdY %d\n",t,tIdX,tIdY);
    if(t != T){   
        grid[3*(N*tIdX +tIdY)+ 2] = grid[3*(N*tIdX +tIdY)+ 1];
        grid[3*(N*tIdX +tIdY)+ 1] = grid[3*(N*tIdX +tIdY)+ 0];
    }
        

}

/**
 * @brief Imprime la matriz, junto con los valores de las iteraciones
 * anteriores
 *
 * @param grid Matriz de flotantes de NxN
 * @param N Tamaño de la matriz
 */
void printGrid(float* grid,int N){
    for(int i = 0;i <N;i++){
        for(int j = 0;j <N;j++){
            printf("%f ",grid[3*(N*i +j)+ 0]);
        }
        printf("\n");
    }
}

/**
 * @brief Libera la memoria de la matriz
 *
 * @param grid Matriz de flotantes de NxN
 * @param N Tamaño de la matriz
 */
void freeGrid(float* grid, int N){
    free(grid);
}

/**
 * @brief Guarda el valor de la matriz en un archivo en formato binario
 *
 * @param grid Matriz de flotantes de NxN
 * @param N Tamaño de la matriz
 * @param filename Nombre del archivo de salida
 */
void save(float* grid,int N,char*filename){
    float *H = (float*)malloc(N*N*sizeof(float));
    for(int i = 0;i <N;i++){
        for(int j = 0;j <N;j++){
            H[i*N+j] = grid[3*(N*i +j)+ 0];
        }
    }
    FILE *f = fopen(filename,"w");
    fwrite(H,sizeof(float),N*N,f);
    fclose(f);
    free(H);
}

__host__ int main(int argc, char *const *argv)
{

    options_t options;
    options.filename = (char *)malloc(15 * sizeof(char));

    if (argc != 11)
    {
        printf("Argumentos Incorrectos\nUSO: ./wave -N tamano_grilla -x tamano_bloque_en_X -y tamano_bloque_en_Y -T numero_de_pasos -f archivo_de_salida\n");
        exit(1);
    }
    int c;

    while (((c = getopt(argc, argv, "N:x:y:T:f:")) != -1))
    {
        switch (c)
        {
        case 'N':
            options.grid_size = atof(optarg);
            break;
        case 'x':
            options.x = atof(optarg);
            break;
        case 'y':
            options.y = atof(optarg);
            break;
        case 'T':
            options.steps = atof(optarg);
            break;
        case 'f':
            options.filename = optarg;
            break;
        default:
            break;
        }
    }

    dim3 blocksize;
    dim3 gridsize;

    gridsize.x = options.grid_size / blocksize.x;
    gridsize.y = options.grid_size / blocksize.y;
    blocksize.x = options.x;
    blocksize.y = options.y;
    
    //printf("N:%d T:%d H:%d f:%s\n", options.grid_size, options.steps, options.threads, options.filename, options.steps);
    float *grid = initializeGrid(options.grid_size);

    float *d_grid;
    hipMalloc((void **) &d_grid, 3*options.grid_size*options.grid_size*sizeof(float));
    hipMemcpy(d_grid, grid, 3 * options.grid_size * options.grid_size * sizeof(float), hipMemcpyHostToDevice);

    for(int t = 0; t <= options.steps; t++){
        schroedinger<<<gridsize,blocksize>>>(d_grid, options.grid_size, options.steps, t);
        hipDeviceSynchronize();
    }
    hipMemcpy(grid, d_grid, 3 * options.grid_size * options.grid_size * sizeof(float), hipMemcpyDeviceToHost);
    //printGrid(grid, options.grid_size);
    
    hipFree(d_grid);
    save(grid, options.grid_size, options.filename);
    freeGrid(grid, options.grid_size);
    
}